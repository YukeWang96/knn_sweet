#include "hip/hip_runtime.h"
#include "hipblas.h"
#include <hip/hip_fp16.h>
#include "hipblas.h"

#include<omp.h>
#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <pthread.h>
#include "common8.h"


// __device__ __forceinline__ int __half2int_rn( __half floatVal ) {
//  int intVal = __float_as_int( floatVal );
//  return (intVal >= 0 ) ? intVal : intVal ^ 0x7FFFFFFF;
// }
// __device__ __forceinline__ __half __int2half_rn( int intVal ) {
//  return __int_as_float( (intVal >= 0) ? intVal : intVal ^ 0x7FFFFFFF);
// }
__device__ __half atomicMin_float(__half *address, __half val){
    int val_int = __half2int_rn(val);
    int old = atomicMin((int *)address, val_int);
    return __int2half_rn(old);
}
__device__ __half atomicMax_float(__half *address, __half val){
    int val_int = __half2int_rn(val);
    int old = atomicMax((int *)address, val_int);
    return __int2half_rn(old);
}
__device__ __half atomicAdd_float(__half *address, __half val){
    int val_int = __half2int_rn(val);
    int old = atomicAdd((int *)address, val_int);
    return __int2half_rn(old);
}

void check(hipError_t status, const char *message){
	if(status != hipSuccess)
		cout <<message<<endl;
}

__global__ void Norm(__half *point, __half *norm, int size, int dim){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid < size){
		__half dist = 0.0f;
		for(int i = 0; i < dim; i++){
			__half tmp = point[tid * dim + i];
			dist += tmp * tmp ;
		}
		norm[tid] = dist; 
	}
}

__global__ void AddAll(__half *queryNorm_dev, __half *repNorm_dev, __half *query2reps_dev, int size, int rep_nb){
	int tx = threadIdx.x + blockIdx.x * blockDim.x;
	int ty = threadIdx.y + blockIdx.y * blockDim.y;
	if(tx < size && ty < rep_nb){
		__half temp = query2reps_dev[ty * size + tx];
		temp += (queryNorm_dev[tx] + repNorm_dev[ty]);
		query2reps_dev[ty * size + tx] = hsqrt(temp);
	}
}
__global__ void findQCluster(__half *query2reps_dev,P2R *q2rep_dev, int size, int rep_nb, __half *maxquery_dev, R2all_static_dev *req2q_static_dev){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < size){
		__half temp = FLT_MAX;
		int index = -1;
		for(int i = 0; i < rep_nb; i++){
			__half tmp = query2reps_dev[i * size+ tid];
			if(temp > tmp){
				index = i;
				temp = tmp;
			}
		}
		q2rep_dev[tid] = {index, temp};
		atomicAdd(&req2q_static_dev[index].npoints,1);
		atomicMax_float(&maxquery_dev[index],temp);
	}
}
__global__ void findTCluster(__half *source2reps_dev,P2R *s2rep_dev, int size, int rep_nb, R2all_static_dev *req2s_static_dev){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < size){
		__half temp = FLT_MAX;
		int index = -1;
		for(int i = 0; i < rep_nb; i++){
			__half tmp = source2reps_dev[i * size+ tid];
			if(temp > tmp){
				index = i;
				temp = tmp;
			}
		}
		s2rep_dev[tid] = {index, temp};
		atomicAdd(&req2s_static_dev[index].npoints,1);
		//atomicMax_float(&maxquery_dev[index],temp);
	}
}
__global__ void fillQMembers(P2R *q2rep_dev, int size, int *repsID, R2all_dyn_p *req2q_dyn_p_dev){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < size){
		int repId = q2rep_dev[tid].repIndex;
		int memberId = atomicAdd(&repsID[repId], 1);
		req2q_dyn_p_dev[repId].memberID[memberId] = tid;
	}
}
__global__ void fillTMembers(P2R *s2rep_dev, int size, int *repsID, R2all_dyn_p *req2s_dyn_p_dev){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < size){
		int repId = s2rep_dev[tid].repIndex;
		int memberId = atomicAdd(&repsID[repId], 1);
		req2s_dyn_p_dev[repId].sortedmembers[memberId] = {tid, s2rep_dev[tid].dist2rep};
	}
}
__device__ int reorder = 0;
__global__ void reorderMembers( int rep_nb, int *repsID, int *reorder_members,R2all_dyn_p *req2q_dyn_p_dev){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < rep_nb){
		if(repsID[tid]!=0){
			int reorderId = atomicAdd(&reorder, repsID[tid]);
		//printf("reorder Id %d %d\n",tid, repsID[tid]);//reorderId);
			memcpy(reorder_members + reorderId, req2q_dyn_p_dev[tid].memberID, repsID[tid]*sizeof(int)) ;
		}
	}
}
__global__ void selectReps_cuda(__half * queries_dev, int query_nb, __half *qreps_dev, int qrep_nb, int *qIndex_dev, int *totalSum_dev, int totalTest, int dim){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid < totalTest * qrep_nb * qrep_nb){
		int test = tid/(qrep_nb*qrep_nb);
		int repId = int (tid%(qrep_nb*qrep_nb))/qrep_nb;
		__half distance = Edistance(queries_dev + qIndex_dev[test * qrep_nb + repId]*dim, queries_dev + qIndex_dev[test*qrep_nb + int (tid%(qrep_nb*qrep_nb))%qrep_nb]*dim, dim);
		// __half distance = 0;
		atomicAdd(&totalSum_dev[test],int(distance));
	}
}
__device__ int repTest = 0;
__global__ void selectReps_max(__half *queries_dev, int query_nb, __half *qreps_dev, int qrep_nb, int *qIndex_dev, int *totalSum_dev, int totalTest, int dim){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid == 0){
		__half distance = 0.0f;
		for(int i = 0; i < totalTest; i++){
			if(distance < __int2half_rn(totalSum_dev[i])){
				//printf("distnace %d\n",totalSum_dev[i]);
				distance = __int2half_rn(totalSum_dev[i]);
				repTest = i;
			}	
		}
		printf("repTest %d\n",repTest);
	}
}
__global__ void selectReps_copy(__half *queries_dev, int query_nb, __half *qreps_dev, int qrep_nb, int *qIndex_dev, int *totalSum_dev, int totalTest, int dim){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid < qrep_nb){
		memcpy(qreps_dev+tid*dim, queries_dev + qIndex_dev[repTest * qrep_nb + tid] * dim, dim * sizeof(__half));
	}
}
void print_last_error()
/* just run hipGetLastError() and print the error message if its return value is not hipSuccess */
{
  hipError_t hipError_t;

  hipError_t = hipGetLastError();
  if(hipError_t != hipSuccess)
  {
    printf("  hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
  }
}
void clusterReps(__half * &queries_dev,
				 __half * &sources_dev, 
				 __half * &qreps_dev, 
				 __half * &sreps_dev, 
				 __half * &maxquery_dev, 
				 P2R * &q2rep_dev, 
				 P2R * &s2rep_dev, 
				 R2all_static_dev * &rep2q_static_dev, 
				 R2all_static_dev * &rep2s_static_dev, 
				 R2all_dyn_p * &rep2q_dyn_p_dev, 
				 R2all_dyn_p * &rep2s_dyn_p_dev, 
				 __half * &query2reps_dev,
				 P2R * &q2rep,     
				 P2R * &s2rep,     
				 R2all_static * &rep2q_static,         
				 R2all_static * &rep2s_static,     
				 R2all_dyn_v * &rep2q_dyn_v,		
				 R2all_dyn_v * &rep2s_dyn_v,     
				 __half * &query2reps,
				R2all_dyn_p * &rep2q_dyn_p, 
				R2all_dyn_p * &rep2s_dyn_p, 
				int * &reorder_members){


	hipMalloc((void **)&query2reps_dev, qrep_nb * query_nb * sizeof(__half));
	hipError_t status;
	status = hipMalloc((void **)&queries_dev, query_nb * dim * sizeof(__half));
	check(status,"Malloc queries failed\n");
	status = hipMemcpy(queries_dev, queries, query_nb * dim * sizeof(__half), hipMemcpyHostToDevice);
	check(status,"Memcpy queries failed\n");


	status = hipMalloc((void **)&sources_dev, source_nb * dim * sizeof(__half));
	check(status,"Malloc sources failed\n");
	status = hipMemcpy(sources_dev, sources, source_nb * dim * sizeof(__half), hipMemcpyHostToDevice);
	check(status,"Mem sources failed\n");

	status = hipMalloc((void **)&qreps_dev, qrep_nb * dim * sizeof(__half));
	check(status,"Malloc reps failed\n");
	//status = hipMemcpy(qreps_dev, qreps, qrep_nb * dim * sizeof(__half), hipMemcpyHostToDevice);
	//check(status,"Mem reps failed\n");

	status = hipMalloc((void **)&sreps_dev, srep_nb * dim * sizeof(__half));
	check(status,"Malloc reps failed\n");
	//status = hipMemcpy(sreps_dev, sreps, srep_nb * dim * sizeof(__half), hipMemcpyHostToDevice);
	//check(status,"Mem reps failed\n");

	int totalTest = 10;
	int *qIndex_dev, *qIndex;
	qIndex = (int *)malloc(totalTest * qrep_nb * sizeof(int));
	hipMalloc((void **)&qIndex_dev, qrep_nb * totalTest * sizeof(int));
	srand(2015);
	for(int i = 0;i<totalTest;i++)
		for(int j =0 ;j<qrep_nb;j++)
			qIndex[i*qrep_nb +j]=rand()%query_nb;
	hipMemcpy(qIndex_dev, qIndex, totalTest * qrep_nb * sizeof(int), hipMemcpyHostToDevice);
	int *totalSum, *totalSum_dev;
	hipMalloc((void **)&totalSum_dev, totalTest * sizeof(__half));
	hipMemset(totalSum_dev, 0, totalTest*sizeof(__half));
	totalSum = (int *)malloc(totalTest*sizeof(__half));

	selectReps_cuda<<<(totalTest*qrep_nb*qrep_nb+255)/256,256>>>(queries_dev, query_nb, qreps_dev, qrep_nb, qIndex_dev, totalSum_dev, totalTest, dim);

	hipDeviceSynchronize();
	print_last_error();


	selectReps_max<<<1,1>>>(queries_dev, query_nb, qreps_dev, qrep_nb, qIndex_dev, totalSum_dev, totalTest, dim);
	selectReps_copy<<<(qrep_nb + 255)/256,256>>>(queries_dev, query_nb, qreps_dev, qrep_nb, qIndex_dev, totalSum_dev, totalTest, dim);



	qIndex = (int *)malloc(totalTest * srep_nb * sizeof(int));
	hipMalloc((void **)&qIndex_dev, srep_nb * totalTest * sizeof(int));

	srand(2015);
	for(int i = 0;i<totalTest;i++)
		for(int j =0 ;j<srep_nb;j++)
			qIndex[i*srep_nb +j]=rand()%source_nb;
	hipMemcpy(qIndex_dev, qIndex, totalTest * srep_nb * sizeof(int), hipMemcpyHostToDevice);


	hipMemset(totalSum_dev, 0, totalTest*sizeof(__half));


	selectReps_cuda<<<(totalTest*srep_nb*srep_nb+255)/256,256>>>(sources_dev, source_nb, sreps_dev, srep_nb, qIndex_dev, totalSum_dev, totalTest, dim);
	selectReps_max<<<1,1>>>(sources_dev, source_nb, sreps_dev, srep_nb, qIndex_dev, totalSum_dev, totalTest, dim);

	selectReps_copy<<<(srep_nb + 255)/256,256>>>(sources_dev, source_nb, sreps_dev, srep_nb, qIndex_dev, totalSum_dev, totalTest, dim);
	hipDeviceSynchronize();	


	hipMalloc((void **)&rep2q_static_dev, qrep_nb * sizeof(R2all_static_dev));
	check(status,"Malloc rep2qs_static failed\n");
	hipMemcpy(rep2q_static_dev, rep2q_static, qrep_nb * sizeof(R2all_static_dev), hipMemcpyHostToDevice);
	check(status,"Memcpy rep2qs_static failed\n");

	hipMalloc((void **)&rep2s_static_dev, srep_nb * sizeof(R2all_static_dev));
	check(status,"Malloc rep2qs_static failed\n");
	hipMemcpy(rep2s_static_dev, rep2s_static, srep_nb * sizeof(R2all_static_dev), hipMemcpyHostToDevice);
	check(status,"Memcpy rep2qs_static failed\n");

	int block = 256;

	__half *queryNorm_dev, *qrepNorm_dev, *sourceNorm_dev, *srepNorm_dev;
	hipMalloc((void **)&queryNorm_dev,query_nb * sizeof(__half));
	hipMalloc((void **)&sourceNorm_dev,source_nb * sizeof(__half));
	hipMalloc((void **)&qrepNorm_dev, qrep_nb * sizeof(__half));
	hipMalloc((void **)&srepNorm_dev, srep_nb * sizeof(__half));


	hipblasHandle_t handle_blas = 0;
	hipblasCreate(&handle_blas);
	__half alpha = -2.0, beta = 0.0;
	const __half* queries_dev_const = &(*queries_dev);
	const __half* qreps_dev_const = &(*qreps_dev);
	const __half* alpha_const = &alpha;
	const __half* beta_const = &beta;

	//hipDeviceSynchronize();
	struct timespec t3,t4,t35;
	timePoint(t3);
	
	// hipblasSgemm('T','N', query_nb, qrep_nb, dim, -2.0, (float*)queries_dev,dim, (float*)qreps_dev,dim, 0.0, (float*)query2reps_dev,query_nb);

	hipblasHgemm(handle_blas, HIPBLAS_OP_T, HIPBLAS_OP_N, query_nb, qrep_nb, dim, alpha_const, queries_dev_const, dim, qreps_dev_const, dim, beta_const, query2reps_dev, query_nb);

	hipDeviceSynchronize();
	timePoint(t35);
	printf("hipblasSgemm warm up time %f\n", timeLen(t3,t35));
	print_last_error();

	timePoint(t1);
	Norm<<<(query_nb + 255)/256,256>>>(queries_dev, queryNorm_dev, query_nb, dim);

	// hipblasSgemm('T','N', query_nb, qrep_nb, dim, (__half)-2.0, queries_dev,dim, qreps_dev,dim,(__half)0.0,query2reps_dev,query_nb);

	hipblasHgemm(handle_blas, HIPBLAS_OP_T, HIPBLAS_OP_N, query_nb, qrep_nb, dim, alpha_const, queries_dev_const, dim, qreps_dev_const, dim, beta_const, query2reps_dev, query_nb);

	hipDeviceSynchronize();
	print_last_error();


	timePoint(t3);
	Norm<<<(qrep_nb + 255)/256, 256>>>(qreps_dev,qrepNorm_dev,qrep_nb,dim);
	dim3 block2D(16,16,1);
	dim3 grid2D_q((query_nb+15)/16,(qrep_nb+15)/16,1);
	AddAll<<<grid2D_q,block2D>>>(queryNorm_dev,qrepNorm_dev,query2reps_dev,query_nb, qrep_nb);
	//hipMemcpy(query2reps, query2reps_dev, rep_nb * query_nb * sizeof(__half), hipMemcpyDeviceToHost);


	hipMalloc((void **)&maxquery_dev, qrep_nb * sizeof(__half));
	hipMemset(maxquery_dev,0,qrep_nb * sizeof(__half));


	status = hipMalloc((void **)&q2rep_dev, query_nb * sizeof(P2R));
	check(status,"Malloc q2rep failed\n");
	findQCluster<<<(query_nb + 255)/256,256>>>(query2reps_dev, q2rep_dev, query_nb, qrep_nb, maxquery_dev,rep2q_static_dev);

	timePoint(t35);
	printf("query rep first part time %f\n",timeLen(t3,t35));

	int  *qrepsID;
	hipMalloc((void **)&qrepsID, qrep_nb * sizeof(int));
	hipMemset(qrepsID, 0, qrep_nb * sizeof(int));
	hipMemcpy(rep2q_static, rep2q_static_dev, qrep_nb * sizeof(R2all_static_dev), hipMemcpyDeviceToHost);
	check(status,"Memcpy rep2qs_static failed\n");
	for(int i = 0; i <qrep_nb; i++){
		hipMalloc((void **)&rep2q_dyn_p[i].replist, srep_nb * sizeof(IndexDist));
		hipMalloc((void **)&rep2q_dyn_p[i].kubound, K * sizeof(__half));
		hipMalloc((void **)&rep2q_dyn_p[i].memberID, rep2q_static[i].npoints * sizeof(int));
	}

	hipMalloc((void **)&rep2q_dyn_p_dev, qrep_nb * sizeof(R2all_dyn_p));
	hipMemcpy(rep2q_dyn_p_dev, rep2q_dyn_p, qrep_nb * sizeof(R2all_dyn_p), hipMemcpyHostToDevice);
	fillQMembers<<<(query_nb + 255)/256,256>>>(q2rep_dev, query_nb, qrepsID, rep2q_dyn_p_dev);


	hipMalloc((void **)&reorder_members, query_nb * sizeof(int));
	
	reorderMembers<<<(qrep_nb + 255)/256,256>>>(qrep_nb, qrepsID, reorder_members, rep2q_dyn_p_dev);

	hipDeviceSynchronize();
	print_last_error();

	timePoint(t4);
	printf("query rep time  %f\n",timeLen(t3,t4));
	__half *source2reps = (__half *)malloc(source_nb * srep_nb * sizeof(__half));
	__half *source2reps_dev;
	hipMalloc((void **)&source2reps_dev,source_nb * srep_nb * sizeof(__half));

	hipDeviceSynchronize();

	const __half* sources_dev_const = &(*sources_dev);
	const __half* sreps_dev_const = &(*sreps_dev);

	timePoint(t3);
	Norm<<<(source_nb + 255)/256,256>>>(sources_dev, sourceNorm_dev, source_nb, dim);
	// hipblasSgemm('T','N', source_nb, srep_nb, dim, (__half)-2.0, sources_dev,dim, sreps_dev,dim,(__half)0.0,source2reps_dev,source_nb);
	
	hipblasHgemm(handle_blas, HIPBLAS_OP_T, HIPBLAS_OP_N, source_nb, srep_nb, dim, alpha_const, sources_dev_const, dim, sreps_dev_const, dim, beta_const, source2reps_dev, source_nb);

	hipDeviceSynchronize();
	timePoint(t35);
	print_last_error();
	printf("source rep first part time %f\n",timeLen(t3,t35));

	Norm<<<(srep_nb + 255)/256, 256>>>(sreps_dev,srepNorm_dev,srep_nb,dim);
	dim3 grid2D_s((source_nb+15)/16,(srep_nb+15)/16,1);
	AddAll<<<grid2D_s,block2D>>>(sourceNorm_dev,srepNorm_dev,source2reps_dev,source_nb, srep_nb);

	
	status = hipMalloc((void **)&s2rep_dev, source_nb * sizeof(P2R));
	check(status,"Malloc s2rep failed\n");
	findTCluster<<<(source_nb + 255)/256,256>>>(source2reps_dev, s2rep_dev, source_nb, srep_nb,rep2s_static_dev);
	int  *srepsID;
	hipMalloc((void **)&srepsID, srep_nb * sizeof(int));
	hipMemset(srepsID, 0, srep_nb * sizeof(int));
	hipMemcpy(rep2s_static, rep2s_static_dev, srep_nb * sizeof(R2all_static_dev), hipMemcpyDeviceToHost);
	for(int i = 0;i<srep_nb;i++){
		hipMalloc((void **)&rep2s_dyn_p[i].sortedmembers, rep2s_static[i].npoints * sizeof(R2all_dyn_p));
	}
	hipMalloc((void **)&rep2s_dyn_p_dev, srep_nb * sizeof(R2all_dyn_p));
	hipMemcpy(rep2s_dyn_p_dev, rep2s_dyn_p, srep_nb * sizeof(R2all_dyn_p),hipMemcpyHostToDevice);
	fillTMembers<<<(source_nb + 255)/256,256>>>(s2rep_dev, source_nb, srepsID, rep2s_dyn_p_dev);

	/*
	hipMemcpy(source2reps, source2reps_dev, srep_nb * source_nb * sizeof(__half), hipMemcpyDeviceToHost);
	for(int i = 0; i < source_nb; i++){
		__half distance = FLT_MAX;
		int repIndex = -1;
		for(int j = 0; j < srep_nb; j++){
			__half len = source2reps[j * source_nb + i];//Edistance(getPoint(sources,i), getPoint(reps,j));
			if(distance > len){
				distance = len;
				repIndex = j;
			}
		}
		s2rep[i].repIndex = repIndex;
		s2rep[i].dist2rep = distance;
		*/
/*
		rep2qs_dyn_v[repIndex].VsortedIndex.push_back(i);
		rep2qs_dyn_v[repIndex].VsortedDist.push_back(distance);
		*/
/*	
		IndexDist temp = {i, distance};
		rep2s_dyn_v[repIndex].Vsortedmembers.push_back(temp);
	
	}
	*/
	timePoint(t3);
	
	//hipStream_t *streamID = (hipStream_t *)malloc(srep_nb * sizeof(hipStream_t));
	// printf("srep_nb: %d\n", srep_nb);
#pragma omp parallel for
	for(int i = 0; i < srep_nb; i++){
		//hipStreamCreate(&streamID[i]);
		if(rep2s_static[i].npoints > 0){

			vector<IndexDist> temp;
			printf("hello-----\n");
			temp.resize(rep2s_static[i].npoints);
			printf("hello22222222222222222222\n");

			// hipMemcpy(&temp[0],rep2s_dyn_p[i].sortedmembers, rep2s_static[i].npoints * sizeof(IndexDist), hipMemcpyDeviceToHost);
			print_last_error();

			sort(temp.begin(),temp.end(),sort_inc());
			printf("hello333333333333333333333333\n");

			// printf("here\n");

			//rep2s_static[i].maxdist = temp[rep2s_static[i].npoints-1].dist;
			//rep2s_static[i].mindist = temp[0].dist;
			hipMemcpy(rep2s_dyn_p[i].sortedmembers, &temp[0], rep2s_static[i].npoints * sizeof(IndexDist), hipMemcpyHostToDevice);

			print_last_error();
#if debug
			cout<<"max "<<rep2qs_static[i].maxsource<<" min: "<<rep2qs_static[i].minsource<<" Qpoints:"<<rep2qs_static[i].noqueries<<" Spoints:"<<rep2qs_static[i].nosources<<endl;
#endif
		}
	}		

	timePoint(t4);
	hipFree(query2reps_dev);
	hipMalloc((void **)&query2reps_dev, query_nb * srep_nb * sizeof(__half));
	dim3 grid2D_qsrep((query_nb+15)/16,(srep_nb+15)/16,1);

	// hipblasSgemm('T','N', query_nb, srep_nb, dim, (__half)-2.0, queries_dev,dim, sreps_dev,dim,(__half)0.0,query2reps_dev,query_nb);
	hipblasHgemm(handle_blas, HIPBLAS_OP_T, HIPBLAS_OP_N, query_nb, srep_nb, dim, alpha_const, queries_dev_const, dim, sreps_dev_const, dim, beta_const, query2reps_dev, query_nb);

	AddAll<<<grid2D_qsrep,block2D>>>(queryNorm_dev,srepNorm_dev,query2reps_dev,query_nb, srep_nb);
	//hipDeviceSynchronize();

	printf("source rep time %f\n",timeLen(t3,t4));
}

void AllocateAndCopyH2D(__half * &queries_dev, __half * &sources_dev, __half * &qreps_dev, __half * &sreps_dev, __half *maxquery_dev,
	P2R * &q2rep_dev, P2R * &s2rep_dev, R2all_static_dev * &rep2q_static_dev, R2all_static_dev * &rep2s_static_dev, R2all_dyn_p * &rep2q_dyn_p_dev, R2all_dyn_p * &rep2s_dyn_p_dev, __half * &query2reps_dev,
	P2R * &q2rep,     P2R * &s2rep,     R2all_static * &rep2q_static,         R2all_static * &rep2s_static,     R2all_dyn_v * &rep2q_dyn_v,		R2all_dyn_v * &rep2s_dyn_v,     __half * &query2reps,
	R2all_dyn_p * &rep2q_dyn_p, R2all_dyn_p * &rep2s_dyn_p){

	hipError_t status;
/*
	status = hipMemcpy(q2rep_dev, q2rep, query_nb * sizeof(P2R), hipMemcpyHostToDevice);
	check(status,"Memcpy reps failed\n");
*/

	//status = hipMemcpy(s2rep_dev, s2rep, source_nb * sizeof(P2R), hipMemcpyHostToDevice);
	//check(status,"Memcpy s2rep failed\n");

	hipMemcpy(rep2q_static_dev, rep2q_static, qrep_nb * sizeof(R2all_static_dev), hipMemcpyHostToDevice);
	check(status,"Memcpy rep2qs_static failed\n");

	hipMemcpy(rep2s_static_dev, rep2s_static, srep_nb * sizeof(R2all_static_dev), hipMemcpyHostToDevice);
	check(status,"Memcpy rep2qs_static failed\n");

	printf("sizeof static static_dev %d %d\n", sizeof(R2all_static), sizeof(R2all_static_dev));


/*

	for(int i = 0; i < srep_nb; i++){
		int nosources_local = rep2s_static[i].npoints;

		if(nosources_local > 0){
			status = hipMalloc((void **)&rep2s_dyn_p[i].sortedmembers, nosources_local * sizeof(IndexDist));
			check(status,"Malloc rep2qs_dyn source failed\n");
			status = hipMemcpy(rep2s_dyn_p[i].sortedmembers, &rep2s_dyn_v[i].Vsortedmembers[0], nosources_local * sizeof(IndexDist), hipMemcpyHostToDevice);
			check(status,"Memcpy rep2qs_dyn source failed\n");

		}
	}

	hipMalloc((void **)&rep2s_dyn_p_dev, srep_nb * sizeof(R2all_dyn_p));
	hipMemcpy(rep2s_dyn_p_dev, rep2s_dyn_p, srep_nb * sizeof(R2all_dyn_p), hipMemcpyHostToDevice);
	*/

}

__global__ void RepsUpperBound(__half *queries_dev, __half *sources_dev, __half *qreps_dev, __half *sreps_dev, __half *query2reps_dev, __half *maxquery_dev,
			P2R *q2rep_dev, P2R *s2rep_dev, R2all_static_dev *rep2q_static_dev, R2all_dyn_p *rep2q_dyn_p_dev,  R2all_static_dev *rep2s_static_dev, R2all_dyn_p *rep2s_dyn_p_dev,
			int query_nb, int source_nb, int qrep_nb, int srep_nb, int dim, int K){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid < qrep_nb){
		//if(fabs(maxquery_dev[tid]-rep2qs_static_dev[tid].maxquery)>0.01)
		//	printf("tid %d %.10f %.10f\n",tid, maxquery_dev[tid],rep2qs_static_dev[tid].maxquery);
		int UBoundCount = 0;
		for(int i = 0; i < srep_nb; i++){
			__half rep2rep = Edistance_128(qreps_dev + tid * dim, sreps_dev + i * dim, dim);
			int count = 0;
			while(count < K && count < rep2s_static_dev[i].npoints){


				//__half g2pUBound = rep2qs_static_dev[tid].maxquery + rep2rep + rep2qs_dyn_p_dev[i].sortedsources[count].dist;
				__half g2pUBound = maxquery_dev[tid] + rep2rep + rep2s_dyn_p_dev[i].sortedmembers[count].dist;

				if(UBoundCount < K){
					rep2q_dyn_p_dev[tid].kubound[UBoundCount] = g2pUBound;
					if(rep2q_static_dev[tid].kuboundMax < g2pUBound)
						rep2q_static_dev[tid].kuboundMax = g2pUBound;

					UBoundCount++;
				}
				else{
					if(rep2q_static_dev[tid].kuboundMax > g2pUBound){
						__half max_local = 0.0f;
						for(int j = 0; j < K; j++){
							if(rep2q_dyn_p_dev[tid].kubound[j]==rep2q_static_dev[tid].kuboundMax){
								rep2q_dyn_p_dev[tid].kubound[j] = g2pUBound;
							}
							if(max_local < rep2q_dyn_p_dev[tid].kubound[j]){
								max_local = rep2q_dyn_p_dev[tid].kubound[j];
							}
						}
						rep2q_static_dev[tid].kuboundMax = max_local;
					}
				}
				count++;
			}
		}
#if debug
		printf("i = %d, %.10f\n",tid,rep2qs_static_dev[tid].kuboundMax);
#endif
	}
}

__global__ void FilterReps
			(__half *queries_dev, __half *sources_dev, __half *qreps_dev, __half *sreps_dev, __half *query2reps_dev, __half *maxquery_dev,
			P2R *q2rep_dev, P2R *s2rep_dev, R2all_static_dev *rep2q_static_dev, R2all_dyn_p *rep2q_dyn_p_dev,  R2all_static_dev *rep2s_static_dev, R2all_dyn_p *rep2s_dyn_p_dev,
			int query_nb, int source_nb, int qrep_nb, int srep_nb, int dim, int K){
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	int tidy = threadIdx.y + blockIdx.y * blockDim.y;   //calculate reps[tidy].replist;
	if(tidx < srep_nb && tidy < qrep_nb){
		__half distance = Edistance(qreps_dev + tidy * dim, sreps_dev + tidx * dim, dim);
		//if(distance - rep2qs_static_dev[tidy].maxquery - rep2qs_static_dev[tidx].maxsource < rep2qs_static_dev[tidy].kuboundMax){
		if(distance - maxquery_dev[tidy] - rep2s_static_dev[tidx].maxdist < rep2q_static_dev[tidy].kuboundMax){

			int rep_id = atomicAdd(&rep2q_static_dev[tidy].noreps,1);
			rep2q_dyn_p_dev[tidy].replist[rep_id].index = tidx;
			rep2q_dyn_p_dev[tidy].replist[rep_id].dist = distance;
#if debug
			printf("tidy = %d tidx = %d distance = %.10f\n", tidy, tidx, distance);
#endif
		}
	}
}


__global__ void NearReps(__half *queries_dev, __half *sources_dev, __half *reps_dev, __half *query2reps_dev,
			P2R *q2rep_dev, P2R *s2rep_dev, R2all_static_dev *rep2qs_static_dev, R2all_dyn_p *rep2qs_dyn_p_dev,
			int query_nb, int source_nb, int rep_nb, int dim, int K){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid < rep_nb){
		IndexDist *temp = rep2qs_dyn_p_dev[tid].replist;
		__half max_local = 0.0f;
		int index = -1;
		for(int i = 0; i < rep2qs_static_dev[tid].noreps; i++){
			if(max_local < temp[i].dist){
				max_local = temp[i].dist;
				index = i;
			}
		}
		IndexDist tmp;
		tmp = temp[index];
		temp[index] = temp[0];
		temp[0] = tmp;
	}
}
__global__ void SortReps(__half *queries_dev, __half *sources_dev, __half *reps_dev, __half *query2reps_dev,
			P2R *q2rep_dev, P2R *s2rep_dev, R2all_static_dev *rep2qs_static_dev, R2all_dyn_p *rep2qs_dyn_p_dev,
			int query_nb, int source_nb, int rep_nb, int dim, int K){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid < rep_nb){
		IndexDist *temp = rep2qs_dyn_p_dev[tid].replist;
		for(int i = 0; i < rep2qs_static_dev[tid].noreps; i++)
			for(int j = i; j < rep2qs_static_dev[tid].noreps; j++){
				if(temp[i].dist > temp[j].dist){
					IndexDist tmp = temp[j];
					temp[j] = temp[i];
					temp[i] = tmp;
				}
			}
	}
				
}
__device__ int Total = 0;
__global__ void printTotal(){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid == 0)
		printf("Total %d\n",Total);
}
__global__ void KNNQuery_base3
			(__half *queries_dev, __half *sources_dev, __half *qreps_dev, __half *sreps_dev, __half *query2reps_dev, __half *maxquery_dev,
			P2R *q2rep_dev, P2R *s2rep_dev, R2all_static_dev *rep2q_static_dev, R2all_dyn_p *rep2q_dyn_p_dev,  R2all_static_dev *rep2s_static_dev, R2all_dyn_p *rep2s_dyn_p_dev,
			int query_nb, int source_nb, int qrep_nb, int srep_nb, int dim, int K, IndexDist * knearest, int * reorder_members){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid < query_nb){
		tid = reorder_members[tid];
		int repIndex = q2rep_dev[tid].repIndex;
		__half theta = rep2q_static_dev[repIndex].kuboundMax;
		int Kcount = 0;
		int count = 0;

		for(int i = 0; i < rep2q_static_dev[repIndex].noreps; i++){
			int minlb_rid = rep2q_dyn_p_dev[repIndex].replist[i].index;
			__half query2rep = 0.0f;
			//if(repIndex != minlb_rid){
				query2rep = query2reps_dev[tid + minlb_rid*query_nb]; 
							//Edistance_128(queries_dev + tid * dim, sreps_dev + minlb_rid * dim, dim);
				atomicAdd(&Total,1);
			//}
			//else
			//	query2rep = q2rep_dev[tid].dist2rep;

			for(int j = rep2s_static_dev[minlb_rid].npoints - 1; j >= 0; j--){


				IndexDist sourcej = rep2s_dyn_p_dev[minlb_rid].sortedmembers[j];
#if debug
				if(tid == 0)
					printf("j %d %.10f\n",sourcej.index,sourcej.dist);
#endif


				__half p2plbound = query2rep - sourcej.dist;
				if(p2plbound > theta)
					break;
				else if(p2plbound < theta * __float2half(-1.0f))
					continue;
				else if(p2plbound <= theta && p2plbound >= theta*__float2half(-1.0f)){
					__half query2source = Edistance_128(queries_dev + tid * dim, sources_dev + sourcej.index * dim, dim);
					count++;
					//atomicAdd(&Total, 1);

#if debug
				if(tid == 0){

					printf("query2source %.10f %.10f %.10f\n", query2source, p2plbound, theta);
				}
#endif

					int insert = -1;
					__half max_local = 0.0f;
					for( int kk = 0; kk < Kcount; kk++){
						if(query2source < knearest[tid * K + kk].dist){
							insert = kk;
							break;
						}
					}
					if(Kcount < K){
						if(insert == -1){
							knearest[tid * K + Kcount] = {sourcej.index, query2source};
						}
						else{
							for(int move = Kcount - 1; move >= insert; move--){
								knearest[tid * K + (move + 1)] = knearest[tid * K + move];
							}
							knearest[tid * K + insert] = {sourcej.index, query2source};
						}
						Kcount++;
					}
					else{  //Kcount = K
						if(insert == -1)
							continue;
						else{
							for(int move = K - 2; move >= insert; move--){
								knearest[tid * K + (move + 1)] = knearest[tid * K + move];
							}

							knearest[tid * K + insert] = {sourcej.index, query2source};
							theta = knearest[(K - 1) + tid * K].dist;
						}

					}
					
				}
			}
		}
		//memcpy(&knearest1[tid * K], knearest, 20 * sizeof(IndexDist));
		/*
		if(tid == 100)
			for(int i = 0; i < K; i++)
				printf("tid i Index Dist %d %d %d %.10f\n",tid, i, knearest[tid * K + i].index, knearest[tid * K +i].dist);
		*/
		
	}
}
__global__ void KNNQuery_base2
			(__half *queries_dev, __half *sources_dev, __half *qreps_dev, __half *sreps_dev, __half *query2reps_dev, __half *maxquery_dev,
			P2R *q2rep_dev, P2R *s2rep_dev, R2all_static_dev *rep2q_static_dev, R2all_dyn_p *rep2q_dyn_p_dev,  R2all_static_dev *rep2s_static_dev, R2all_dyn_p *rep2s_dyn_p_dev,
			int query_nb, int source_nb, int qrep_nb, int srep_nb, int dim, int K, IndexDist * knearest, int * reorder_members){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid < query_nb){
		tid = reorder_members[tid];
		int repIndex = q2rep_dev[tid].repIndex;
		__half theta = rep2q_static_dev[repIndex].kuboundMax;
		int Kcount = 0;
		int count = 0;

		for(int i = 0; i < rep2q_static_dev[repIndex].noreps; i++){
			int minlb_rid = rep2q_dyn_p_dev[repIndex].replist[i].index;
			__half query2rep = 0.0f;
			//if(repIndex != minlb_rid){
				query2rep = query2reps_dev[tid + minlb_rid*query_nb]; 
							//Edistance_128(queries_dev + tid * dim, sreps_dev + minlb_rid * dim, dim);
				atomicAdd(&Total,1);
			//}
			//else
			//	query2rep = q2rep_dev[tid].dist2rep;

			for(int j = rep2s_static_dev[minlb_rid].npoints - 1; j >= 0; j--){


				IndexDist sourcej = rep2s_dyn_p_dev[minlb_rid].sortedmembers[j];
#if debug
				if(tid == 0)
					printf("j %d %.10f\n",sourcej.index,sourcej.dist);
#endif


				__half p2plbound = query2rep - sourcej.dist;
				if(p2plbound > theta)
					break;
				else if(p2plbound < theta*__float2half(-1.0f))
					continue;
				else if(p2plbound <= theta && p2plbound >= theta*__float2half(-1.0f)){
					__half query2source = Edistance_128(queries_dev + tid * dim, sources_dev + sourcej.index * dim, dim);
					count++;
					//atomicAdd(&Total, 1);

#if debug
				if(tid == 0){

					printf("query2source %.10f %.10f %.10f\n", query2source, p2plbound, theta);
				}
#endif

					int insert = -1;
					__half max_local = 0.0f;
					for( int kk = 0; kk < Kcount; kk++){
						if(query2source < knearest[tid + kk * query_nb].dist){
							insert = kk;
							break;
						}
					}
					if(Kcount < K){
						if(insert == -1){
							knearest[tid + Kcount * query_nb] = {sourcej.index, query2source};
						}
						else{
							for(int move = Kcount - 1; move >= insert; move--){
								knearest[tid + (move + 1) * query_nb] = knearest[tid + move * query_nb];
							}
							knearest[tid + insert * query_nb] = {sourcej.index, query2source};
						}
						Kcount++;
					}
					else{  //Kcount = K
						if(insert == -1)
							continue;
						else{
							for(int move = K - 2; move >= insert; move--){
								knearest[tid + (move + 1)*query_nb] = knearest[tid + move*query_nb];
							}

							knearest[tid + insert * query_nb] = {sourcej.index, query2source};
							theta = knearest[(K - 1)*query_nb + tid].dist;
						}

					}
				}
			}
		}
		//memcpy(&knearest1[tid * K], knearest, 20 * sizeof(IndexDist));
		/*
		if(tid == 100)
			for(int i = 0; i < K; i++)
				printf("tid i Index Dist %d %d %d %.10f\n",tid, i, knearest[tid * K + i].index, knearest[tid * K +i].dist);
		*/
		
	}
}
__global__ void KNNQuery_base
			(__half *queries_dev, __half *sources_dev, __half *qreps_dev, __half *sreps_dev, __half *query2reps_dev, __half *maxquery_dev,
			P2R *q2rep_dev, P2R *s2rep_dev, R2all_static_dev *rep2q_static_dev, R2all_dyn_p *rep2q_dyn_p_dev,  R2all_static_dev *rep2s_static_dev, R2all_dyn_p *rep2s_dyn_p_dev,
			int query_nb, int source_nb, int qrep_nb, int srep_nb, int dim, int K, IndexDist * knearest1, int * reorder_members){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid < query_nb){
		tid = reorder_members[tid];
		int repIndex = q2rep_dev[tid].repIndex;
		__half theta = rep2q_static_dev[repIndex].kuboundMax;
		int Kcount = 0;
		int count = 0;

		IndexDist knearest[1000];
		for(int i = 0; i < rep2q_static_dev[repIndex].noreps; i++){
			int minlb_rid = rep2q_dyn_p_dev[repIndex].replist[i].index;
			__half query2rep = 0.0f;
			//if(repIndex != minlb_rid){
				query2rep = query2reps_dev[tid + minlb_rid*query_nb]; 
							//Edistance_128(queries_dev + tid * dim, sreps_dev + minlb_rid * dim, dim);
		 	//	atomicAdd(&Total,1);
			//}
			//else
			//	query2rep = q2rep_dev[tid].dist2rep;

			for(int j = rep2s_static_dev[minlb_rid].npoints - 1; j >= 0; j--){


				IndexDist sourcej = rep2s_dyn_p_dev[minlb_rid].sortedmembers[j];
#if debug
				if(tid == 0)
					printf("j %d %.10f\n",sourcej.index,sourcej.dist);
#endif


				__half p2plbound = query2rep - sourcej.dist;
				if(p2plbound > theta)
					break;
				else if(p2plbound < theta*__float2half(-1.0f))
					continue;
				else if(p2plbound <= theta && p2plbound >= theta*__float2half(-1.0f)){
					__half query2source = Edistance_128(queries_dev + tid * dim, sources_dev + sourcej.index * dim, dim);
					count++;
					atomicAdd(&Total, 1);

#if debug
				if(tid == 0){

					printf("query2source %.10f %.10f %.10f\n", query2source, p2plbound, theta);
				}
#endif

					int insert = -1;
					__half max_local = 0.0f;
					for( int kk = 0; kk < Kcount; kk++){
						if(query2source < knearest[kk].dist){
							insert = kk;
							break;
						}
					}
					if(Kcount < K){
						if(insert == -1){
							knearest[Kcount] = {sourcej.index, query2source};
						}
						else{
							for(int move = Kcount - 1; move >= insert; move--){
								knearest[move + 1] = knearest[move];
							}
							knearest[insert] = {sourcej.index, query2source};
						}
						Kcount++;
					}
					else{  //Kcount = K
						if(insert == -1)
							continue;
						else{
							for(int move = K - 2; move >= insert; move--){
								knearest[move + 1] = knearest[move];
							}

							knearest[insert] = {sourcej.index, query2source};
							theta = knearest[K - 1].dist;
						}

					}
				}
			}
		}
		memcpy(&knearest1[tid * K], knearest, K * sizeof(IndexDist));
		
		/*
		if(tid == 100)
			for(int i = 0; i < K; i++)
				printf("tid i Index Dist %d %d %d %.10f\n",tid, i, knearest[tid * K + i].index, knearest[tid * K +i].dist);
		*/
		
	}
}
__global__ void KNNQuery_theta
			(__half *queries_dev, __half *sources_dev, __half *qreps_dev, __half *sreps_dev, __half *query2reps_dev, __half *maxquery_dev,
			P2R *q2rep_dev, P2R *s2rep_dev, R2all_static_dev *rep2q_static_dev, R2all_dyn_p *rep2q_dyn_p_dev,  R2all_static_dev *rep2s_static_dev, R2all_dyn_p *rep2s_dyn_p_dev,
			int query_nb, int source_nb, int qrep_nb, int srep_nb, int dim, int K, IndexDist * knearest, __half * thetas){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid < query_nb){
		int repIndex = q2rep_dev[tid].repIndex;
		thetas[tid] = rep2q_static_dev[repIndex].kuboundMax;
	}
}
__global__ void KNNQuery
			(__half *queries_dev, __half *sources_dev, __half *qreps_dev, __half *sreps_dev, __half *query2reps_dev, __half *maxquery_dev,
			P2R *q2rep_dev, P2R *s2rep_dev, R2all_static_dev *rep2q_static_dev, R2all_dyn_p *rep2q_dyn_p_dev,  R2all_static_dev *rep2s_static_dev, R2all_dyn_p *rep2s_dyn_p_dev,
			int query_nb, int source_nb, int qrep_nb, int srep_nb, int dim, int K, IndexDist * knearest, __half * thetas, int tpq, int *reorder_members ){
	int ttid = threadIdx.x + blockIdx.x * blockDim.x;
	int tp = ttid % tpq;
	int tid = ttid / tpq;
	if(tid < query_nb){
		tid = reorder_members[tid];
		ttid = tid * tpq + tp;
		int repIndex = q2rep_dev[tid].repIndex;
		//__half theta = rep2q_static_dev[repIndex].kuboundMax;
		int Kcount = 0;
		int count = 0;

		for(int i = 0; i < rep2q_static_dev[repIndex].noreps; i++){
			int minlb_rid = rep2q_dyn_p_dev[repIndex].replist[i].index;
			__half query2rep = 0.0f;
			//if(repIndex != minlb_rid){
				query2rep = //query2reps_dev[tid + minlb_rid*query_nb]; 
							Edistance_128(queries_dev + tid * dim, sreps_dev + minlb_rid * dim, dim);
				//atomicAdd(&Total,1);

			for(int j = rep2s_static_dev[minlb_rid].npoints - 1 - tp; j >= 0; j-=tpq){
				IndexDist sourcej = rep2s_dyn_p_dev[minlb_rid].sortedmembers[j];
#if debug
				if(tid == 0)
					printf("j %d %.10f\n",sourcej.index,sourcej.dist);
#endif
				__half p2plbound = query2rep - sourcej.dist;
				if(p2plbound > thetas[tid])
					break;
				else if(p2plbound < thetas[tid] * __float2half(-1.0f))
					continue;
				else if(p2plbound <= thetas[tid] && p2plbound >= thetas[tid]*__float2half(-1.0f)){
					__half query2source = Edistance_128(queries_dev + tid * dim, sources_dev + sourcej.index * dim, dim);
					count++;
					atomicAdd(&Total, 1);

#if debug
				if(tid == 0){

					printf("query2source %.10f %.10f %.10f\n", query2source, p2plbound, theta);
				}
#endif

					int insert = -1;
					__half max_local = 0.0f;
					for( int kk = 0; kk < Kcount; kk++){
						if(query2source < knearest[ttid * K + kk].dist){
							insert = kk;
							break;
						}
					}
					if(Kcount < K){
						if(insert == -1){
							knearest[ttid * K + Kcount] = {sourcej.index, query2source};
						}
						else{
							for(int move = Kcount - 1; move >= insert; move--){
								knearest[ttid * K + move + 1] = knearest[ttid * K + move];
							}
							knearest[ttid * K + insert] = {sourcej.index, query2source};
						}
						Kcount++;
					}
					else{  //Kcount = K
						if(insert == -1)
							continue;
						else{
							for(int move = K - 2; move >= insert; move--){
								knearest[ttid * K + move + 1] = knearest[ttid * K + move];
							}

							knearest[ttid * K + insert] = {sourcej.index, query2source};
							atomicMin_float(&thetas[tid], knearest[ttid * K + K - 1].dist);
						}

					}
				}
			}
		}
		/*
		if(tid == 100)
			for(int i = 0; i < K; i++)
				printf("tid i Index Dist %d %d %d %.10f\n",tid, i, knearest[tid * K + i].index, knearest[tid * K +i].dist);
				*/
		
	}
}

__global__ void final(int k, IndexDist * knearest, int tpq, int query_nb, IndexDist *final_knearest, int *tag_base){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int *tag = tid * tpq + tag_base;
	if(tid < query_nb){
	for(int i = 0; i < k; i++){
		__half min = knearest[tid * tpq * k + tag[0]].dist;
		int index = 0;
		for(int j = 1; j < tpq ; j++){
			__half value = knearest[(tid * tpq + j)* k + tag[j]].dist;
			if(min > value){
				min = value;
				index = j;
			}
		}
		//if(tid ==100) printf("final i index tag %d %d %d %f\n",i, index, tag[14],knearest[(tid * tpq + 14)* k + tag[14]].dist);
		final_knearest[tid * k + i] = knearest[(tid * tpq + index)* k + tag[index]];
		tag[index]++;
	}
	}
}
void *work(void *para){
	hipFree(0);
}
int main(int argc, char *argv[]){
	pthread_t thread2;
	timePoint(t1);
	int rc = pthread_create(&thread2,NULL, work, NULL);
	//hipFree(0);


	if(argc<9){
		cout<<"usage: ./exe query_nb source_nb dimension rep_nb k input1 input2\n";
		exit(1);
	}

	query_nb = atoi(argv[1]);
	source_nb = atoi(argv[2]);
	dim = atoi(argv[3]);
	qrep_nb = atoi(argv[4]);
	srep_nb = atoi(argv[5]);
	K = atoi(argv[6]);
	char *query_data = argv[7];
	char *source_data = argv[8];

	sources = (__half *)malloc(source_nb * dim * sizeof(__half));
	queries = (__half *)malloc(query_nb * dim * sizeof(__half));

	//Setup for source and query points.
	pointSetup(query_data, source_data);

	qreps = (__half *)malloc(qrep_nb * dim * sizeof(__half));
	sreps = (__half *)malloc(srep_nb * dim * sizeof(__half));
	P2R *q2rep = (P2R *)malloc(query_nb * sizeof(P2R));
	P2R *s2rep = (P2R *)malloc(source_nb * sizeof(P2R));
	R2all_static *rep2q_static = (R2all_static*)malloc(qrep_nb * sizeof(R2all_static));
	R2all_static *rep2s_static = (R2all_static*)malloc(srep_nb * sizeof(R2all_static));
	R2all_dyn_v *rep2q_dyn_v = (R2all_dyn_v *)malloc(qrep_nb * sizeof(R2all_dyn_v));
	R2all_dyn_v *rep2s_dyn_v = (R2all_dyn_v *)malloc(srep_nb * sizeof(R2all_dyn_v));

	__half *query2reps = (__half *)malloc(query_nb * qrep_nb * sizeof(__half));
	__half *queries_dev, *sources_dev, *qreps_dev, *sreps_dev;
	P2R *q2rep_dev, *s2rep_dev;
	R2all_static_dev *rep2q_static_dev;
	R2all_dyn_p *rep2q_dyn_p_dev;
	R2all_static_dev *rep2s_static_dev;
	R2all_dyn_p *rep2s_dyn_p_dev;
	__half *query2reps_dev;
	__half *maxquery_dev;

	int *reorder_members;

	R2all_dyn_p *rep2q_dyn_p = (R2all_dyn_p *)malloc(qrep_nb * sizeof(R2all_dyn_p));
	R2all_dyn_p *rep2s_dyn_p = (R2all_dyn_p *)malloc(srep_nb * sizeof(R2all_dyn_p));
	//Select reps
	timePoint(t1);
	//selectReps(queries, query_nb, qreps, qrep_nb);
	//selectReps(sources, source_nb, sreps, srep_nb);
	//cluster queries and sources to reps
	 hipMalloc((void **)&query2reps_dev, qrep_nb * query_nb * sizeof(__half));
	//timePoint(t1);
	timePoint(t2);
	printf("hipFree time %f\n",timeLen(t1,t2));
	clusterReps(queries_dev, sources_dev, qreps_dev, sreps_dev, maxquery_dev,
				q2rep_dev, s2rep_dev, rep2q_static_dev, rep2s_static_dev, rep2q_dyn_p_dev, rep2s_dyn_p_dev, query2reps_dev,
				q2rep,     s2rep,     rep2q_static,     rep2s_static,     rep2q_dyn_v,     rep2s_dyn_v,     query2reps,		rep2q_dyn_p, rep2s_dyn_p, reorder_members);
	
	
	//tranfer data structures to GPU.
	AllocateAndCopyH2D(queries_dev, sources_dev, qreps_dev, sreps_dev, maxquery_dev,
				q2rep_dev, s2rep_dev, rep2q_static_dev, rep2s_static_dev, rep2q_dyn_p_dev, rep2s_dyn_p_dev, query2reps_dev,
				q2rep,     s2rep,     rep2q_static,     rep2s_static,     rep2q_dyn_v,     rep2s_dyn_v,     query2reps,		rep2q_dyn_p, rep2s_dyn_p);
	timePoint(t2);
	printf("prepo time %f\n",timeLen(t1,t2));
    if(hipGetLastError()!=hipSuccess) cout <<"error 16"<<endl;

	//Kernel 1: upperbound for each rep
	//timePoint(t1);
	RepsUpperBound<<<(qrep_nb+255)/256, 256>>>
											(queries_dev, sources_dev, qreps_dev, sreps_dev, query2reps_dev, maxquery_dev,\
											q2rep_dev, s2rep_dev, rep2q_static_dev, rep2q_dyn_p_dev, rep2s_static_dev, rep2s_dyn_p_dev, \
											query_nb, source_nb, qrep_nb, srep_nb, dim, K);

    if(hipGetLastError()!=hipSuccess) cout <<"Kernel RepsUpperBound failed"<<endl;

	//Kernel 2: filter reps	based on upperbound and lowerbound;
	dim3 block(16,16,1);
	dim3 grid((srep_nb+block.x-1)/block.x, (qrep_nb+block.y-1)/block.y,1);
	FilterReps<<<grid, block>>>
								(queries_dev, sources_dev, qreps_dev, sreps_dev, query2reps_dev, maxquery_dev,\
											q2rep_dev, s2rep_dev, rep2q_static_dev, rep2q_dyn_p_dev, rep2s_static_dev, rep2s_dyn_p_dev, \
											query_nb, source_nb, qrep_nb, srep_nb, dim, K);
						
					


	struct timespec sort_start, sort_end;
	timePoint(sort_start);	
	hipMemcpy(rep2q_static, rep2q_static_dev, qrep_nb * sizeof(R2all_static_dev), hipMemcpyDeviceToHost);
	
#pragma omp parallel for
	for(int i = 0; i < qrep_nb; i++){
		//printf("replist len %d\n",rep2q_static[i].noreps);
		//IndexDist *tmp = (IndexDist *)malloc(rep2qs_static[i].noreps*sizeof(IndexDist));
		vector<IndexDist> temp;
		temp.resize( rep2q_static[i].noreps);
		hipMemcpy(&temp[0], rep2q_dyn_p[i].replist, rep2q_static[i].noreps * sizeof(IndexDist), hipMemcpyDeviceToHost);
		sort(temp.begin(),temp.end(),sort_inc());

		hipMemcpy(rep2q_dyn_p[i].replist, &temp[0], rep2q_static[i].noreps * sizeof(IndexDist), hipMemcpyHostToDevice);

	}
	
	timePoint(sort_end);
	printf("sort query replist time %f\n",timeLen(sort_start,sort_end));


	//SortReps<<<(rep_nb + 127) / 128, 128>>>(queries_dev, sources_dev, reps_dev, query2reps_dev,\
											q2rep_dev, s2rep_dev, rep2qs_static_dev, rep2qs_dyn_p_dev, \
											query_nb, source_nb, rep_nb, dim, K);

	//Kernel 3: knn for each point
	IndexDist * knearest, *final_knearest;
	int tpq = (2048*13)/query_nb;
	IndexDist * knearest_h = (IndexDist *)malloc(query_nb * K * sizeof(IndexDist));
	hipMalloc((void **)&knearest, query_nb * (tpq+1) * K * sizeof(IndexDist));
	int avg_query_nb = int(query_nb / qrep_nb);
	if(tpq>1){
		__half *theta;
		hipMalloc((void **)&theta, query_nb * sizeof(__half));
		KNNQuery_theta<<<(query_nb+255)/256, 256>>>
								(queries_dev, sources_dev, qreps_dev, sreps_dev, query2reps_dev, maxquery_dev,\
											q2rep_dev, s2rep_dev, rep2q_static_dev, rep2q_dyn_p_dev, rep2s_static_dev, rep2s_dyn_p_dev, \
											query_nb, source_nb, qrep_nb, srep_nb, dim, K, knearest, theta);
	//hipMemset(theta, 0, query_nb * sizeof(__half));
		hipDeviceSetCacheConfig(hipFuncCachePreferL1);

		KNNQuery<<<(tpq*query_nb+255)/256, 256>>>
								(queries_dev, sources_dev, qreps_dev, sreps_dev, query2reps_dev, maxquery_dev,\
											q2rep_dev, s2rep_dev, rep2q_static_dev, rep2q_dyn_p_dev, rep2s_static_dev, rep2s_dyn_p_dev, \
											query_nb, source_nb, qrep_nb, srep_nb, dim, K, knearest, theta, tpq, reorder_members);
		final_knearest = knearest + query_nb * tpq * K;

		int * tag_base;
		hipMalloc((void **)&tag_base, tpq * query_nb * sizeof(int));
		hipMemset(tag_base, 0, tpq * query_nb * sizeof(int));
		final<<<(query_nb+255)/256,256>>>(K, knearest,tpq, query_nb, final_knearest, tag_base);
	}
	else{
		KNNQuery_base<<<(query_nb +255)/256, 256>>>
								(queries_dev, sources_dev, qreps_dev, sreps_dev, query2reps_dev, maxquery_dev,\
											q2rep_dev, s2rep_dev, rep2q_static_dev, rep2q_dyn_p_dev, rep2s_static_dev, rep2s_dyn_p_dev, \
											query_nb, source_nb, qrep_nb, srep_nb, dim, K, knearest, reorder_members);
	}
	hipDeviceSynchronize();
	timePoint(t2);
	printf("total time %f\n",timeLen(t1,t2));
	printTotal<<<1,1>>>();
	if(tpq>1)
		hipMemcpy(knearest_h, final_knearest, query_nb * K * sizeof(IndexDist),hipMemcpyDeviceToHost);
	else
		hipMemcpy(knearest_h, knearest, query_nb * K * sizeof(IndexDist),hipMemcpyDeviceToHost);
	
	int i = 100;
		for(int j=0;j<K;j++)
			printf("i,k %d %d  %d %f\n",i,j, knearest_h[i*K+j].index,knearest_h[i*K+j].dist);
	/*
	for(int i =0 ;i < 1000;i++)
		for(int j=0;j<K;j++)
			printf("i,k %d %d  %d %f\n",i,j, knearest_h[i*K+j].index,knearest_h[i*K+j].dist);*/
	hipDeviceSynchronize();

	//R2	
	free(queries);
	free(sources);
	free(qreps);
	free(sreps);
	free(q2rep);
	free(s2rep);
	free(rep2q_static);
	free(rep2q_dyn_v);
	free(rep2s_static);
	free(rep2s_dyn_v);
	return 0;
}
